#include "hip/hip_runtime.h"
#include "structs.cuh"
#include "init.h"
#include "LBMconsts.cuh"
#include "phys.h"
#include <nvfunctional>

#include "materials.cuh"

template<class F> __global__ void fill(F);

void init(){
  parsHost.iStep=0;
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

  printf("Malloc data\n");
  parsHost.data.malloc_data(Nx,Ny,Nz);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

  cuTimer init_timer;
  fill<<<dim3(Nx,Ny),Nz>>>( [] __device__(int ix, int iy,int iz) {return blank_mat(ix,iy,iz);} );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  printf("\n");

  printf("Initialization time: %.2f ms\n", init_timer.gettime());
  
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

}

template<class F> __global__ void fill(F func){
  /*int ix = threadIdx.x+blockIdx.x*blockDim.x;
  int iy = threadIdx.y+blockIdx.y*blockDim.y;
  int iz = threadIdx.z+blockIdx.z*blockDim.z;*/
  int ix = blockIdx.x;
  int iy = blockIdx.y;
  int iz = threadIdx.x;
  Cell c;
  std::pair<ftype, ftype4> rho_uT = func(ix,iy,iz);

  const ftype rho = rho_uT.first;
  const ftype3 vel = make_ftype3(rho_uT.second.x, rho_uT.second.y, rho_uT.second.z) ;
  const ftype T = rho_uT.second.w;

  assert(rho_uT.second.w>LBMconsts::Tmin);
  assert(rho_uT.second.w<LBMconsts::Tmax);

  ftype feq[LBMconsts::Qn];
  c.calcEq(feq, rho_uT.first, make_ftype3(0,0,0), rho_uT.second.w );
  for(int iq=0; iq<LBMconsts::Qn; iq++) c.f[iq]=feq[iq];
  c.rho = rho;
  c.vel = vel;
  c.T = T;

  pars.data.set_cell(c, 0, ix,iy,iz);
  pars.data.set_cell(c, 1, ix,iy,iz);
}

