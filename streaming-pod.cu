#include "hip/hip_runtime.h"
#include "momentsMatrix.cuh"
struct InterpolateStruct{
  ftype3 shifts;
  int3 stencilMinPos;
  __device__ __forceinline__ InterpolateStruct(ftype3 xf, int3 _sminpos): stencilMinPos(_sminpos) {
    shifts = xf - make_ftype3(stencilMinPos);
  }
  template<class T,class F> __device__ inline T calc(F);
};

template<int RegOrder=-1> __global__ __launch_bounds__(LBMconsts::Qn) void  streaming_collision(int ibn) {
  ibn+= blockIdx.x;
  if(ibn>=Nx*Ny*Nz) return;

  const int ix = ibn%Nx;
  const int iy = ibn/Nx%Ny;
  const int iz = ibn/(Nx*Ny);

  const int ild = 0;
  const int ist = 1;

  Cell cell = pars.data.get_cell(ild, ix,iy,iz);

  const int3 ic = make_int3(ix, iy, iz);

  const int3 Nxyz = make_int3(Nx,Ny,Nz);
  using namespace LBMconsts;

  int useSHmem4momMatrix = 0;
  if(sizeof(MomentsMatrix)<48*1024) useSHmem4momMatrix = 1;

  /*if(blockIdx.x==0) {
  MomentsMatrix mtest;
  for(int i=0;i<Qn;i++) for(int j=0;j<Qn;j++) mtest.m[i][j] = 2.+sqrt(abs(sin(100*i)*cos(i*j)));//1.23456+1.23*i/(j+1)+j/(i+1)*0.21;
  MomentsMatrix mtestorig=mtest;
  mtest.inverse();
  ftype mmul[Qn][Qn];
  for(int i=0;i<Qn;i++) for(int j=0;j<Qn;j++) {
    ftype sum=0;
    for(int k=0;k<Qn;k++) sum+= mtestorig.m[i][k]*mtest.m[k][j+Qn];
    if(abs(sum)<1e-8) sum=0;
    printf("%g ",sum);
    if(j==Qn-1) printf("\n");
  }
  assert(0);
  }*/

  MomentsMatrix* Mm,*Mm0;
  __shared__ MomentsMatrix _mmsh;
  __shared__ MomentsMatrix* shMm;
  //__shared__ MomentsMatrix _mmsh0;
  //__shared__ MomentsMatrix* shMm0;

  if(useSHmem4momMatrix) {
    Mm = &_mmsh;
    //Mm0 = &_mmsh0;
  } else {
    if(threadIdx.x==0) {
      shMm = (MomentsMatrix*)malloc(sizeof(MomentsMatrix));
      //shMm0 = (MomentsMatrix*)malloc(sizeof(MomentsMatrix));
      assert(shMm);
      //assert(shMm0);
    }
    __syncthreads();
    Mm=shMm;
    //Mm0=shMm0;
  }

  //cell.vel*=0; cell.T=PPdev.initial.T0/10.0;
  __shared__ Cell cell_new;
  int Niter=0;
  while(Niter<100) {
    ftype T = cell.T;
    ftype rho=cell.rho;
    ftype3 vel = cell.vel;

    ftype4 gauge = make_ftype4(vel.x, vel.y, vel.z, sqrt(T/TLat));

    __syncthreads();
    Mm->init(gauge);
    __syncthreads();
    Mm->inverse();
    __syncthreads();
    /* Mm0->init(make_ftype4(0,0,0,1));
      __syncthreads();
      Mm0->inverse();
      __syncthreads(); */

    const int iq = threadIdx.x;
    ftype3 v = ef[iq]*gauge.w + make_ftype3(gauge.x,gauge.y,gauge.z);

    int3 interpStencilMinPos = ic - make_int3(PPdev.stencilInterpWidth/2);
    const ftype3 xf = make_ftype3(ic)-v;
    if(PPdev.stencilFixed==0) {
      ftype3 pos = xf-make_ftype3(0.5*PPdev.stencilInterpWidth);
      interpStencilMinPos = make_int3( round(pos.x), round(pos.y), round(pos.z) );
    }
    if(DIM<2) interpStencilMinPos.y = ic.y;
    if(DIM<3) interpStencilMinPos.z = ic.z;

    InterpolateStruct interpolate(xf, interpStencilMinPos);
    //cell_new.f[iq] = pars.data.tiles[ild][interpStencilMinPos.x+interpStencilMinPos.y+interpStencilMinPos.z].f[iq];

    if(RegOrder<0) {
      cell_new.f[iq] = interpolate.calc<ftype>( [&] __device__ (int index) {
        ftype mVec[Qn];

        ftype4 igauge = pars.data.tiles[ild][index].uT[0];
        igauge.w = sqrt(igauge.w/TLat);
        calc_moments_vec( igauge, pars.data.tiles[ild][index].f, mVec );

        /*ftype all_fi[Qn]; for(int ii=0;ii<Qn;ii++) all_fi[ii] = Mm->get_inv(iq,mVec);
         calc_moments_vec( make_ftype4(0,0,0,1), all_fi, mVec );
         for(int ii=0;ii<Qn;ii++) if(MomentsPower[ii].x+MomentsPower[ii].y+MomentsPower[ii].z>4) mVec[ii]=0;
         const ftype fi_reg = Mm0->get_inv(iq,mVec);
         return fi_reg;*/

        const ftype fi = Mm->get_inv(iq, mVec);
        return fi;
        //return pars.data.tiles[ild][index].f[iq];
      } );
    } else {
      TensorCoeffs<RegOrder> an = interpolate.calc< TensorCoeffs<RegOrder> >( [&] __device__ (int index) {
        TensorCoeffs<RegOrder> an_p;
      
        ftype4 igauge = pars.data.tiles[ild][index].uT[0];
        igauge.w = sqrt(igauge.w/TLat);
      
        calc_moments_tensors( igauge, pars.data.tiles[ild][index].f, an_p);
        return an_p;
      } );

      TensorCoeffs<RegOrder> dn =  convertAtoD(an, gauge);
      cell_new.f[iq] = eval_fi_Hermit(dn, iq);

      /*ftype4 tmpgauge = pars.data.tiles[ild][174].uT[0];
      tmpgauge.w = sqrt(tmpgauge.w/TLat);
      ftype mVec[Qn];
      calc_moments_vec( tmpgauge, pars.data.tiles[ild][174].f, mVec );
      if(ix==174 && iq==0) printf("Niter=%d moments=(%g %g %g %g %g %g)\n     An=(%g %g %g %g %g %g)\n      Dn=(%g %g %g %g %g %g)\n", 
                                    Niter, mVec[0],mVec[1],mVec[2],mVec[3],mVec[4],mVec[5],
                                    an.k[0],an.k[1],an.k[2],an.k[3],an.k[4],an.k[5],
                                    dn.k[0],dn.k[1],dn.k[2],dn.k[3],dn.k[4],dn.k[5]
                             );*/



    }
    __syncthreads();
    if(threadIdx.x==0) {
      ftype4 Vrho = make_ftype4(0,0,0,0);
      ftype M2 = 0;
      for(int ik=0; ik<Qn; ik++) {
        ftype3 v_k = ef[ik]*gauge.w + make_ftype3(gauge.x,gauge.y,gauge.z);
        Vrho+= make_ftype4(v_k.x,v_k.y,v_k.z,1)*cell_new.f[ik];
        M2+= dot(v_k,v_k)*cell_new.f[ik];
      }
      cell_new.rho = Vrho.w;
      cell_new.vel = make_ftype3(Vrho.x,Vrho.y,Vrho.z)/cell_new.rho;
      cell_new.T = M2/cell_new.rho-dot(cell_new.vel,cell_new.vel); cell_new.T/=DIM;
      if(PPdev.fixedTemperature) cell_new.T=cell.T;
      if(cell_new.T<0) {
        printf("Convergency problem: cell %d %d %d (iteration %d) got negative T=%g, reset to positive\n",
                                          ix,iy,iz,Niter, cell_new.T );
        cell_new.T=-cell_new.T;
      }
    }
    __syncthreads();

    Niter++;
    if( isConv(cell,cell_new) ) { cell=cell_new; break; }
    cell=cell_new;
    __syncthreads();
  }
  //printf("ixyz=%d %d %d Niter=%d\n",ix,iy,iz, Niter);
  __syncthreads();
  if(threadIdx.x==0) {
    if(!useSHmem4momMatrix) free(Mm);

    ftype feq[Qn];
    Cell::calcEq(feq, cell.rho, make_ftype3(0,0,0), TLat);
    collision(cell.f,feq);

    pars.data.set_cell(cell, ist, ix,iy,iz);

    pars.data.tiles[ist][ix+iy*Nx+iz*Nx*Ny].Niter[0] = Niter;
  }
}

inline __device__ ftype LagrPol(int ix,int iy,int iz, const ftype3 shifts, const int N);

template<class Interp_t, class F> __device__ inline Interp_t InterpolateStruct::calc(F func) {
  const int3 Nxyz = make_int3(Nx,Ny,Nz);
  Interp_t val(0);
  const int Npoints = PPdev.stencilInterpWidth+1;
  for(int xs=0; xs<Npoints; xs++) {
    for(int ys=0; ys<((DIM<2)?1:Npoints); ys++) {
      for(int zs=0; zs<((DIM<3)?1:Npoints); zs++) {
        const int3 crd = ( stencilMinPos+make_int3(xs,ys,zs)+Nxyz )%Nxyz;
        const int index = crd.x + crd.y*Nx + crd.z*Nx*Ny;
        const ftype coeff = LagrPol(xs,ys,zs, shifts, Npoints);
        auto Tcoffsp = func(index);
        Tcoffsp*= coeff;
        val+= Tcoffsp;
      }
    }
  }
  return val;
}
inline __device__ ftype LagrPol(int ix,int iy,int iz, const ftype3 shifts, const int N){
  ftype a=1;
  if(DIM>0) for(int ixp=0; ixp<N; ixp++) if(ixp!=ix) a*= (shifts.x-ixp)/(ix-ixp);
  if(DIM>1) for(int iyp=0; iyp<N; iyp++) if(iyp!=iy) a*= (shifts.y-iyp)/(iy-iyp);
  if(DIM>2) for(int izp=0; izp<N; izp++) if(izp!=iz) a*= (shifts.z-izp)/(iz-izp);
  return a;
}
