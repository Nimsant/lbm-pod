#include "hip/hip_runtime.h"

template<int RO=-1> __global__ __launch_bounds__(LBMconsts::Qn) void  streaming_collision(int ibn) {
  if(threadIdx.x!=0) return;

  /*const int ix = blockIdx.x*CUDAstep::Nb.x + threadIdx.x;
  const int iy = blockIdx.y*CUDAstep::Nb.y + threadIdx.y;
  const int iz = blockIdx.z*CUDAstep::Nb.z + threadIdx.z;*/

  ibn+= blockIdx.x;
  if(ibn>=Nx*Ny*Nz) return;

  const int ix = ibn%Nx;
  const int iy = ibn/Nx%Ny;
  const int iz = ibn/(Nx*Ny);

  const int ild = 0;
  const int ist = 1;

  Cell cell = pars.data.get_cell(ild, ix,iy,iz);
  ftype rho=cell.rho;
  ftype3 vel = cell.vel;

  const int3 ic = make_int3(ix, iy, iz);

  const int3 Nxyz = make_int3(Nx,Ny,Nz);
  using namespace LBMconsts;

  ftype fnew[Qn], feq[Qn];
  ftype4 Vrho = make_ftype4(0,0,0,0);
  ftype T=0;

  for(int iq=0; iq<Qn; iq++) {
    const int3 icn = (ic-e[iq]+Nxyz)%Nxyz;
    const int nind = icn.x + icn.y*Nx + icn.z*Nx*Ny;
    fnew[iq] = pars.data.tiles[ild][nind].f[iq];
    Vrho+= make_ftype4(e[iq].x,e[iq].y,e[iq].z,1)*fnew[iq];
    T+= dot(e[iq],e[iq])*fnew[iq];
  }

  const ftype3 Vel = make_ftype3(Vrho.x,Vrho.y,Vrho.z)/Vrho.w;
  T = T/Vrho.w - dot(Vel,Vel);
  T/= DIM;

  Cell::calcEq(feq, Vrho.w, Vel, T);
  collision(fnew,feq);

  cell.rho = 0;
  cell.vel = make_ftype3(0,0,0);
  cell.T = 0;
  for(int iq=0; iq<Qn; iq++) {
    cell.f[iq] = fnew[iq];
    cell.rho+= fnew[iq];
    ftype3 ef = make_ftype3(e[iq]);
    cell.vel+= ef*fnew[iq];
    cell.T+= dot(ef,ef)*fnew[iq];
  }
  cell.vel/= cell.rho;
  cell.T = cell.T/cell.rho - dot(cell.vel,cell.vel);
  cell.T/= DIM;

  pars.data.set_cell(cell, ist, ix,iy,iz);
}
