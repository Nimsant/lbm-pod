#include "hip/hip_runtime.h"
#include "structs.cuh"
#include "semaphore.h"
#include "LBMconsts.cuh"
#include "phys.h"

#include "collision.cu"
#include "streaming.cu"

void calcLBM(int it, std::vector<double>& timings);
void simple_drop();
void debug_print();

struct FullIntegrals{
  double mass;
  double3 momentum;
  double Energy, kinEn, Enstropy, Entropy;
  long long int MaxVelocity;
};
__managed__ FullIntegrals TotMoments;
__global__ void total_moments( FullIntegrals& totM );

void calcStep(int REV=1){
  cuTimer calct;
  parsHost.iStep++;
  std::vector<double> timings;
  int Ntiles=0;
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  calcLBM(parsHost.iStep, timings);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  double phys_time=parsHost.iStep;
  double calc_time = calct.gettime();
  printf("Step %6d (physical time %6.3f ms) | Performance: %.2f ms (%.2f MLU/sec) | timings: ", 
      parsHost.iStep ,phys_time, calc_time,
      Nx*Ny*Nz/calc_time*1e-3     );
  for(auto tmg: timings) printf("%.2f ",tmg);
  printf("\n");
  
  if(parsHost.iStep%PPhost.StepIterPeriod==0) {
  memset( &TotMoments, 0, sizeof(FullIntegrals) );
  total_moments<<<dim3(Nx,Ny),Nz>>>(TotMoments); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  printf("Total Conservations: Mass %.15f Momentum( %.12f %.12f %.12f ), M2 %.12f\n",
                  TotMoments.mass, TotMoments.momentum.x, TotMoments.momentum.y, TotMoments.momentum.z, TotMoments.Energy );
  printf("Total Characteristics: KineticEnergy: %.15f Enstropy: %.15f Entropy: %.15f MaxVelocity: %.15f\n",
                  TotMoments.kinEn, TotMoments.Enstropy, TotMoments.Entropy, *((double*)&(TotMoments.MaxVelocity)) );
  }
}

template<int n> struct KerRunner {
  static void run() { streaming_collision <n> <<<Nx*Ny*Nz,LBMconsts::Qn>>>(0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() ); }
};
 
void calcLBM(int it, std::vector<double>& timings){
  cuTimer t0; double new_time=0, prev_time=0;

  hipDeviceProp_t prop;
  CHECK_ERROR( hipGetDeviceProperties( &prop, 0) );
  const int MaxBlocksPerSM=32;
  if(it==1) {
    printf("GPU SM count %3d \n", prop.multiProcessorCount);
    //CHECK_ERROR( hipDeviceSetLimit(hipLimitStackSize, 256*1024) );
    CHECK_ERROR( hipDeviceSetLimit( hipLimitMallocHeapSize, MaxBlocksPerSM*prop.multiProcessorCount*sizeof(MomentsMatrix) ) );
  }

  TemplateSwitcher<5, KerRunner<5> >::run( PPhost.RegOrder );
  //streaming_collision<<<Nx*Ny*Nz,1>>>(0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  //using namespace CUDAstep;
  /*for(int ibn=0; ibn<Nx*Ny*Nz; ibn+= MaxBlocksPerSM*prop.multiProcessorCount) {
    printf("step %5d progress = %6d/%6d\r", it, ibn, Nx*Ny*Nz); fflush(stdout);
    streaming_collision<<<MaxBlocksPerSM*prop.multiProcessorCount,1>>>(ibn);
    //streaming_collision<<<1,1>>>(ibn);
    hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  }*/
  parsHost.data.swap_ptrs();
  copy2dev( parsHost, pars );
  debug_print(); timings.push_back( t0.getlaptime() );

  debug_print(); timings.push_back( t0.getlaptime() );

}

__global__ void total_moments( FullIntegrals& totMom ){
  const int ix=blockIdx.x;
  const int iy=blockIdx.y;
  const int iz=threadIdx.x;
  const int index = ix+iy*Nx+iz*Nx*Ny;
  using namespace LBMconsts;

  Cell cell = pars.data.get_cell(0, ix,iy,iz);
  const ftype rho=cell.rho;
  const ftype3 vel = cell.vel;
  const ftype T = cell.T;
  ftype entropy = 0; for(int iq=0;iq<LBMconsts::Qn;iq++) entropy+= cell.f[iq]*log(cell.f[iq]/w[iq]);
  Cell cellMx = pars.data.get_cell(0, (ix-1+Nx)%Nx, iy, iz);
  Cell cellPx = pars.data.get_cell(0, (ix+1   )%Nx, iy, iz);
  Cell cellMy = pars.data.get_cell(0, ix, (iy-1+Ny)%Ny, iz);
  Cell cellPy = pars.data.get_cell(0, ix, (iy+1   )%Ny, iz);
  Cell cellMz = pars.data.get_cell(0, ix, iy, (iz-1+Nz)%Nz);
  Cell cellPz = pars.data.get_cell(0, ix, iy, (iz+1   )%Nz);
  ftype3 vorticity;
  vorticity.x = 0.5*(cellPy.vel.z-cellMy.vel.z) - 0.5*(cellPz.vel.y-cellMz.vel.y);
  vorticity.y = 0.5*(cellPz.vel.x-cellMz.vel.x) - 0.5*(cellPx.vel.z-cellMx.vel.z);
  vorticity.z = 0.5*(cellPx.vel.y-cellMx.vel.y) - 0.5*(cellPy.vel.x-cellMy.vel.x);

  atomicAdd(&totMom.mass      , rho);
  atomicAdd(&totMom.momentum.x, rho*vel.x );
  atomicAdd(&totMom.momentum.y, rho*vel.y );
  atomicAdd(&totMom.momentum.z, rho*vel.z );
  atomicAdd(&totMom.Energy    , rho*T*DIM/2 + rho*dot(vel,vel)/2 );
  atomicAdd(&totMom.kinEn     , rho*dot(vel,vel)/2 );
  atomicAdd(&totMom.Enstropy  , rho*dot(vorticity,vorticity)/2 );
  atomicAdd(&totMom.Entropy   , entropy );
  atomicMax(&totMom.MaxVelocity, __double_as_longlong(sqrt(dot(vel,vel))) );
}

inline void debug_print(){
   return;
}
