#include "hip/hip_runtime.h"
#include "structs.cuh"
#include "semaphore.h"
#include "LBMconsts.cuh"
#include "phys.h"

#include "collision.cu"
#include "streaming.cu"

void calcLBM(int it, std::vector<double>& timings);
void simple_drop();
void debug_print();

__managed__ double TotMoments[5];
__global__ void total_moments( double totM[5] );

void calcStep(int REV=1){
  cuTimer calct;
  parsHost.iStep++;
  std::vector<double> timings;
  int Ntiles=0;
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  calcLBM(parsHost.iStep, timings);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  double phys_time=parsHost.iStep;
  double calc_time = calct.gettime();
  printf("Step %6d (physical time %6.3f ms) | Performance: %.2f ms (%.2f MLU/sec) | timings: ", 
      parsHost.iStep ,phys_time, calc_time,
      Nx*Ny*Nz/calc_time*1e-3     );
  for(auto tmg: timings) printf("%.2f ",tmg);
  printf("\n");
  
  memset( TotMoments, 0, sizeof(TotMoments) );
  total_moments<<<dim3(Nx,Ny),Nz>>>(TotMoments); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  printf("Total Moments: Mass %.15f Momentum( %.12f %.12f %.12f ), M2 %.12f\n", TotMoments[0], TotMoments[1], TotMoments[2], TotMoments[3], TotMoments[4] );
}

template<int n> struct KerRunner {
  static void run() { streaming_collision <n> <<<Nx*Ny*Nz,LBMconsts::Qn>>>(0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() ); }
};
 
void calcLBM(int it, std::vector<double>& timings){
  cuTimer t0; double new_time=0, prev_time=0;

  hipDeviceProp_t prop;
  CHECK_ERROR( hipGetDeviceProperties( &prop, 0) );
  const int MaxBlocksPerSM=32;
  if(it==1) {
    printf("GPU SM count %3d \n", prop.multiProcessorCount);
    //CHECK_ERROR( hipDeviceSetLimit(hipLimitStackSize, 256*1024) );
    CHECK_ERROR( hipDeviceSetLimit( hipLimitMallocHeapSize, MaxBlocksPerSM*prop.multiProcessorCount*sizeof(MomentsMatrix) ) );
  }

  TemplateSwitcher<5, KerRunner<5> >::run( PPhost.RegOrder );
  //streaming_collision<<<Nx*Ny*Nz,1>>>(0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  //using namespace CUDAstep;
  /*for(int ibn=0; ibn<Nx*Ny*Nz; ibn+= MaxBlocksPerSM*prop.multiProcessorCount) {
    printf("step %5d progress = %6d/%6d\r", it, ibn, Nx*Ny*Nz); fflush(stdout);
    streaming_collision<<<MaxBlocksPerSM*prop.multiProcessorCount,1>>>(ibn);
    //streaming_collision<<<1,1>>>(ibn);
    hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  }*/
  parsHost.data.swap_ptrs();
  copy2dev( parsHost, pars );
  debug_print(); timings.push_back( t0.getlaptime() );

  debug_print(); timings.push_back( t0.getlaptime() );

}

__global__ void total_moments( double totMom[5] ){
  const int ix=blockIdx.x;
  const int iy=blockIdx.y;
  const int iz=threadIdx.x;
  const int index = ix+iy*Nx+iz*Nx*Ny;
  using namespace LBMconsts;

  Cell cell = pars.data.get_cell(0, ix,iy,iz);
  const ftype rho=cell.rho;
  const ftype3 vel = cell.vel;
  const ftype T = cell.T;

  atomicAdd(&totMom[0], rho);
  atomicAdd(&totMom[1], rho*vel.x );
  atomicAdd(&totMom[2], rho*vel.y );
  atomicAdd(&totMom[3], rho*vel.z );
  atomicAdd(&totMom[4], rho*T*DIM + rho*dot(vel,vel) );
}

inline void debug_print(){
   return;
}
