#include "hip/hip_runtime.h"
#include "cuda_math.h"
#include <stdio.h>
#include <stdlib.h>
#include "structs.cuh"

#include "im2D.h"
#include "im3D.hpp"

#include "LBMconsts.cuh"

#include "phys.h"

im3D_pars im3DHost;
void calcStep(int REV=1);

const char* FuncStr[] = {
  "rho", "Vx", "Vy", "Vz", "T", "Press", "Vorticity", "Niter"
};

__global__ void __launch_bounds__(Nz) draw(float* buf) {
  int iz=threadIdx.x;
  int ix=blockIdx.x;
  int iy=blockIdx.y;

  float* pbuf=&buf[ix+gridDim.x*(iy+gridDim.y*iz)];
  Cell cell = pars.data.get_cell(0, ix,iy,iz);
  Cell cellPx = pars.data.get_cell(0, (ix+1   )%Nx,iy,iz ), cellPy = pars.data.get_cell(0, ix, (iy+1   )%Ny,iz ), cellPz = pars.data.get_cell(0, ix, iy, (iz+1   )%Nz );
  Cell cellMx = pars.data.get_cell(0, (ix-1+Nx)%Nx,iy,iz ), cellMy = pars.data.get_cell(0, ix, (iy-1+Ny)%Ny,iz ), cellMz = pars.data.get_cell(0, ix, iy, (iz-1+Nz)%Nz );
  ftype rho=cell.rho;
  ftype3 vel = cell.vel;
  ftype3 vort;
  vort.x = 0.5*(cellPy.vel.z-cellMy.vel.z) - 0.5*(cellPz.vel.y-cellMz.vel.y);
  vort.y = 0.5*(cellPz.vel.x-cellMz.vel.x) - 0.5*(cellPx.vel.z-cellMx.vel.z);
  vort.z = 0.5*(cellPx.vel.y-cellMx.vel.y) - 0.5*(cellPy.vel.x-cellMy.vel.x);
  const int MaxFunc = sizeof(FuncStr)/sizeof(char*);
  
  switch(pars.nFunc) {
      case MaxFunc-1: *pbuf=float(cell.Niter); break;
      case 0 : *pbuf=float(rho); break;
      case 1 : *pbuf=float(vel.x); break;
      case 2 : *pbuf=float(vel.y); break;
      case 3 : *pbuf=float(vel.z); break;
      case 4 : *pbuf=float(cell.T); break;
      case 5 : *pbuf=float(rho*cell.T); break;
      case 6 : *pbuf=float(length(vort)); break;
      default: break;
  }
}
void draw_all(){
  CHECK_ERROR( hipMemset(parsHost.arr4im.Arr3Dbuf,0,((long long int)parsHost.arr4im.Nx)*parsHost.arr4im.Ny*parsHost.arr4im.Nz*sizeof(float)) );
  draw<<<dim3(parsHost.arr4im.Nx,parsHost.arr4im.Ny),parsHost.arr4im.Nz>>>(parsHost.arr4im.Arr3Dbuf);
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  im3DHost.initCuda(parsHost.arr4im);
}

void idle_func_calc::step() {
  for(int ii=0;ii<PPhost.StepIterPeriod;ii++) {
    calcStep(); 
    t++;
  }
  draw_all();
  im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod);
  recalc_at_once=true;
}

static void key_func(unsigned char key, int x, int y) {
  if(type_diag_flag>=2) printf("keyN=%d, coors=(%d,%d)\n", key, x, y);
  if(key == 'h') {
    printf("\
======= Управление:\n\
  <¦>  \tИзменение функции для визуализации: Values_level543210¦isBnd_level543210¦AMR_LEVEL\n\
«Enter»\tПересчёт одного шага\n\
   b   \tвключает пересчёт в динамике (см. «Управление динамикой»)\n\
"); im3DHost.print_help();
    return;
  }
  ftype t0;
  switch(key) {
  //case '>': if(parsHost.nFunc<parsHost.MaxFunc) parsHost.nFunc++; break;
  //case '<': if(parsHost.nFunc>0) parsHost.nFunc--; break;
  case '>': parsHost.nFunc = (parsHost.nFunc+1)%parsHost.MaxFunc; break;
  case '<': parsHost.nFunc = (parsHost.nFunc+parsHost.MaxFunc-1)%parsHost.MaxFunc; break;
  case 13: for(int ii=0;ii<PPhost.StepIterPeriod;ii++) calcStep(+1); /*im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod)*/; break;
  case 8 : for(int ii=0;ii<PPhost.StepIterPeriod;ii++) calcStep(-1); /*im3DHost.save_png(parsHost.iStep/PPhost.StepIterPeriod)*/; break;
  default: if(!im3DHost.key_func(key, x, y)) {
  if(type_diag_flag>=0) printf("По клавише %d в позиции (%d,%d) нет никакого действия\n", key, x, y);
  } return;
  }
  copy2dev( parsHost, pars );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  draw_all();
  recalc_at_once=true;
}
static void draw_func() {
  im3DHost.fName = FuncStr[parsHost.nFunc]; 
  im2D.draw(im3DHost.reset_title()); 
}

static void idle_func() { im3DHost.recalc_func(); }
static void mouse_func(int button, int state, int x, int y) { im3DHost.mouse_func(button, state, x, y); }
static void motion_func(int x, int y) { im3DHost.motion_func(x, y); }
static void special_func(int key, int x, int y) { 
  im3DHost.special_func(key, x, y);
  if(key == GLUT_KEY_F2) {
//    parsHost.drawArrows^=1;
    copy2dev( parsHost, pars ); draw_all();
    recalc_at_once=true;
  }
}

int print_help() {
  printf("help | using in test|batch mode:\n ./lbm [--help|--test|--batch]\n");
  printf("using in interactive mode:\n ./lbm %s\n", im3DHost.command_line_help_string());
  im3DHost.print_command_line_help();
  return 0;
}
void read_float3(float* v, char* str);
float read_float(char* str);

void launch_im3D(int argc, char** argv){
  parsHost.nFunc = 0; parsHost.MaxFunc = sizeof(FuncStr)/sizeof(char*);
    
  cudaTimer tm; tm.start();
  parsHost.reset_im();
  im3DHost.reset(parsHost.arr4im);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );
  im2D.get_device(3,0);
  im2D.init_image(argc,argv, im3DHost.bNx, im3DHost.bNy, "im3D");
  im3DHost.init3D(parsHost.arr4im); im3DHost.iz0=parsHost.arr4im.Nx-1; im3DHost.key_func('b',0,0);
  im3DHost.initCuda(parsHost.arr4im);
  draw_all();

  if(type_diag_flag>=1) printf("Настройка GLUT и запуск интерфейса\n");
  glutIdleFunc(idle_func);
  glutKeyboardFunc(key_func);
  glutMouseFunc(mouse_func);
  glutMotionFunc(motion_func);
  glutDisplayFunc(draw_func);
  glutSpecialFunc(special_func);
  if(type_diag_flag>=0) printf("Init cuda device: %.1f msec\n", tm.stop());
  glutMainLoop();
}

float get_val_from_arr3D(int ix, int iy, int iz) {
  Arr3D_pars& arr=parsHost.arr4im;
  if(arr.inCPUmem) return arr.Arr3Dbuf[arr.get_ind(ix,iy,iz)];
  float res=0.0;
  if(arr.inGPUmem) CHECK_ERROR(hipMemcpy(&res, arr.get_ptr(ix,iy,iz), sizeof(float), hipMemcpyDeviceToHost));
  return res;
}

