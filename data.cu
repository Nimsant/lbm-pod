#include "data.cuh"

void Data_t::malloc_data(const int Nx, const int Ny, const int Nz){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  printf("Total data size = %g GB\n",double(sz)/1024/1024/1024); 
  for(int ipar:{0,1} ) CHECK_ERROR( hipMalloc((void**)&tiles[ipar], sz ) );
  CHECK_ERROR( hipHostMalloc((void**)&tilesHost, sz ) );
  for(auto itile: tiles ) CHECK_ERROR( hipMemset(itile, 0, sz ) );
  CHECK_ERROR( hipMemset(tilesHost, 0, sz ) );
};
void Data_t::copyHost2Dev(){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  for(auto itiles: tiles ) CHECK_ERROR( hipMemcpy(itiles, tilesHost, sz, hipMemcpyHostToDevice ) );
}
void Data_t::copyDev2Host(const int ipar){
  const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
  const size_t sz = long(Nx)*Ny*Nz/Ns3*sizeof(Tile);
  CHECK_ERROR( hipMemcpy(tilesHost, tiles[ipar], sz, hipMemcpyDeviceToHost ) );
}

void Data_t::swap_ptrs(){
  Tile* tmpptr = tiles[0];
  tiles[0] = tiles[1];
  tiles[1] = tmpptr;
}

