#include "hip/hip_runtime.h"
__host__ __device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  const ftype T0=cs2;
  #ifdef NON_ISOTHERMAL_RELAXATION
  Tcur=Tempr;
  #endif

  const ftype dT2 = dT*dT;
  const ftype dT4 = dT2*dT2;
  
  const ftype u2 = dot(u,u);
  const ftype u4 = u2*u2;
  const int TERM1 = (EqOrder>=1);
  const int TERM2 = (EqOrder>=2);
  #ifdef NON_ISOTHERMAL_RELAXATION
  const int TERM3 = 1;//(EqOrder>=3);
  const int TERM4 = 1;//(EqOrder>=4);
  #else
  const int TERM3 = 0;
  const int TERM4 = 0;
  #endif
  const ftype mxwU = 1 - TERM2*u2*0.5*dT;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype ei2 = dot(eidx,eidx);
    const ftype ei4 = ei2*ei2;
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    const ftype eu4 = eu2*eu2;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tcur-T0)*0.5*dT*(ei2*dT-DIM)+
                 TERM3*1./6.*eu*dT*( eu2*dT2-3*u2*dT + 3*(Tcur-T0)*dT*(ei2*dT-DIM-2) )+
                 TERM4*1./24.*dT4*( eu4 + 3*T0*T0*u4 - 6*T0*eu2*u2 + 6*(Tcur-T0)*eu2*ei2 + 3*(Tcur-T0)*(Tcur-T0)*ei4
                                    - 6*T0*(Tcur-T0)*(Tcur-T0)*(DIM+2)*ei2 + 3*T0*T0*(Tcur-T0)*(Tcur-T0)*DIM*(DIM+2)
                                    - 6*T0*(Tcur-T0)*u2*ei2 - 6*T0*(Tcur-T0)*(DIM+4)*eu2 + 6*T0*T0*(Tcur-T0)*(DIM+2)*u2
                                  );
    feq[i] = w[i]*rho*mxw;
  }
}
/*__host__ __device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  #ifdef NON_ISOTHERMAL_RELAXATION
  Tcur=Tempr
  #endif

  const ftype dT2 = dT*dT;
  const ftype dT3 = dT*dT*dT;
  const ftype dT4 = dT*dT*dT*dT;
  const ftype dT5 = dT*dT*dT*dT*dT;
  
  const ftype u2 = dot(u,u);
  const ftype u4 = u2*u2;
  const int TERM1 = (EqOrder>=1);
  const int TERM2 = (EqOrder>=2);
  const int TERM3 = (EqOrder>=3);
  const int TERM4 = (EqOrder>=4);
  const int TERM5 = (EqOrder>=5);
  const ftype mxwU = 1 - TERM2*u2*0.5*dT + TERM4*u4*0.125*dT2;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    const ftype eu3 = eu*eu*eu;
    const ftype eu4 = eu*eu*eu*eu;
    const ftype eu5 = eu*eu*eu*eu*eu;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tempr-cs2)*0.5*dT*(dot(eidx,eidx)*dT-DIM) +
                 TERM3*eu3*ftype(1./6.)*dT3  - TERM3*eu*u2*0.5*dT2 +
                 TERM4*eu4*ftype(1./24)*dT4  - TERM4*eu2*u2*0.25*dT3 +
                 TERM5*eu5*ftype(1./120)*dT5 - TERM5*eu3*u2*ftype(1./12)*dT4 + TERM5*eu*u4*0.125*dT3;
    feq[i] = w[i]*rho*mxw;
    #ifdef NON_ISOTHERMAL_RELAXATION
    feq[i] = w_get(i,Tempr)*rho*mxw;
    #endif
  }
}*/

inline __host__ __device__ void Data_t::set_cell(const Cell& c, const int ipar, const int ix, const int iy, const int iz){
  static_assert(Tile::Ns==1);
  for(int iq=0; iq<Cell::Qn; iq++) {
    const int3 gCrd = make_int3(ix, iy, iz);
    const int gInd =  gCrd.x + gCrd.y*Nx+ gCrd.z*Nx*Ny;

    if(Tile::Ns==1) {
      tiles[ipar][gInd].f[iq] = c.f[iq];
      tiles[ipar][gInd].uT[0] = make_ftype4(c.vel.x, c.vel.y, c.vel.z, c.T);
    } else {
      Tile* ctile = &tiles[ipar][ gCrd.x/Tile::Ns + gCrd.y/Tile::Ns*(Nx/Tile::Ns) + gCrd.z/Tile::Ns*(Nx/Tile::Ns)*(Ny/Tile::Ns) ];
      const int3 intileCrd = gCrd%Tile::Ns;
      const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
      ctile->f[iq + (intileCrd.x+intileCrd.y*Tile::Ns+intileCrd.z*Tile::Ns*Tile::Ns)*Ns3 ] = c.f[iq];
    }
  }
}
