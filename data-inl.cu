#include "hip/hip_runtime.h"
__host__ __device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  #ifdef NON_ISOTHERMAL_RELAXATION
  Tcur=Tempr;
  #endif

  const ftype dT2 = dT*dT;
  
  const ftype u2 = dot(u,u);
  const int TERM1 = (EqOrder>=1);
  const int TERM2 = (EqOrder>=2);
  const ftype mxwU = 1 - TERM2*u2*0.5*dT;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tcur-cs2)*0.5*dT*(dot(eidx,eidx)*dT-DIM);
    feq[i] = w[i]*rho*mxw;
  }
}
/*__host__ __device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  #ifdef NON_ISOTHERMAL_RELAXATION
  Tcur=Tempr
  #endif

  const ftype dT2 = dT*dT;
  const ftype dT3 = dT*dT*dT;
  const ftype dT4 = dT*dT*dT*dT;
  const ftype dT5 = dT*dT*dT*dT*dT;
  
  const ftype u2 = dot(u,u);
  const ftype u4 = u2*u2;
  const int TERM1 = (EqOrder>=1);
  const int TERM2 = (EqOrder>=2);
  const int TERM3 = (EqOrder>=3);
  const int TERM4 = (EqOrder>=4);
  const int TERM5 = (EqOrder>=5);
  const ftype mxwU = 1 - TERM2*u2*0.5*dT + TERM4*u4*0.125*dT2;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    const ftype eu3 = eu*eu*eu;
    const ftype eu4 = eu*eu*eu*eu;
    const ftype eu5 = eu*eu*eu*eu*eu;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tempr-cs2)*0.5*dT*(dot(eidx,eidx)*dT-DIM) +
                 TERM3*eu3*ftype(1./6.)*dT3  - TERM3*eu*u2*0.5*dT2 +
                 TERM4*eu4*ftype(1./24)*dT4  - TERM4*eu2*u2*0.25*dT3 +
                 TERM5*eu5*ftype(1./120)*dT5 - TERM5*eu3*u2*ftype(1./12)*dT4 + TERM5*eu*u4*0.125*dT3;
    feq[i] = w[i]*rho*mxw;
    #ifdef NON_ISOTHERMAL_RELAXATION
    feq[i] = w_get(i,Tempr)*rho*mxw;
    #endif
  }
}*/

inline __host__ __device__ void Data_t::set_cell(const Cell& c, const int ipar, const int ix, const int iy, const int iz){
  static_assert(Tile::Ns==1);
  for(int iq=0; iq<Cell::Qn; iq++) {
    const int3 gCrd = make_int3(ix, iy, iz);
    const int gInd =  gCrd.x + gCrd.y*Nx+ gCrd.z*Nx*Ny;

    if(Tile::Ns==1) {
      tiles[ipar][gInd].f[iq] = c.f[iq];
      tiles[ipar][gInd].uT[0] = make_ftype4(c.vel.x, c.vel.y, c.vel.z, c.T);
    } else {
      Tile* ctile = &tiles[ipar][ gCrd.x/Tile::Ns + gCrd.y/Tile::Ns*(Nx/Tile::Ns) + gCrd.z/Tile::Ns*(Nx/Tile::Ns)*(Ny/Tile::Ns) ];
      const int3 intileCrd = gCrd%Tile::Ns;
      const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
      ctile->f[iq + (intileCrd.x+intileCrd.y*Tile::Ns+intileCrd.z*Tile::Ns*Tile::Ns)*Ns3 ] = c.f[iq];
    }
  }
}
